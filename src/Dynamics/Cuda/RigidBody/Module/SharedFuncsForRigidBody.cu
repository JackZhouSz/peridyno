#include "hip/hip_runtime.h"
#include "SharedFuncsForRigidBody.h"

namespace dyno
{
	__global__ void SF_ApplyTransform(
		DArrayList<Transform3f> instanceTransform,
		const DArray<Vec3f> diff,
		const DArray<Vec3f> translate,
		const DArray<Mat3f> rotation,
		const DArray<Mat3f> rotationInit,
		const DArray<Pair<uint, uint>> binding)
	{
		int tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= rotation.size())
			return;

		Pair<uint, uint> pair = binding[tId];

		Transform3f ti = Transform3f(translate[tId] + diff[tId], rotation[tId] * rotationInit[tId].transpose());

		instanceTransform[pair.first][pair.second] = ti;
	}

	void ApplyTransform(
		DArrayList<Transform3f>& instanceTransform, 
		const DArray<Vec3f>& diff,
		const DArray<Vec3f>& translate,
		const DArray<Mat3f>& rotation,
		const DArray<Mat3f>& rotationInit,
		const DArray<Pair<uint, uint>>& binding)
	{
		cuExecute(rotation.size(),
			SF_ApplyTransform,
			instanceTransform,
			diff,
			translate,
			rotation,
			rotationInit,
			binding);

	}
}