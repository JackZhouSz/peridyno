#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "DamplingParticleIntegrator.h"
#include "Node.h"
#include "SceneGraphFactory.h"

namespace dyno
{
	//IMPLEMENT_TCLASS(DamplingParticleIntegrator, TDataType)

	template<typename TDataType>
	DamplingParticleIntegrator<TDataType>::DamplingParticleIntegrator()
		:ComputeModule()
	{
		this->inAttribute()->tagOptional(true);
	}

	template<typename TDataType>
	void DamplingParticleIntegrator<TDataType>::begin()
	{
	}


	template<typename Real, typename Coord>
	__global__ void K_Disspation_Velocity(
		DArray<Coord> vel,
		Real disspation)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vel.size()) return;

		vel[pId] *= disspation;
	}

	template<typename TDataType>
	void DamplingParticleIntegrator<TDataType>::end()
	{
		int vNum = this->inPosition()->getData().size();
		
		cuExecute(vNum,
			K_Disspation_Velocity,
			this->inVelocity()->getData(),
			this->inAirDisspation()->getData());
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DArray<Coord> vel,
		DArray<Coord> contactForce,
		DArray<Coord> Norm,
		Coord gravity,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vel.size()) return;

		vel[pId] += dt * (gravity);
	}



	template<typename Real, typename Coord>
	__global__ void K_UpdateVelocity(
		DArray<Coord> vel,
		DArray<Coord> pos,
		DArray<Coord> contactForce,
		DArray<Coord> Norm,
		DArray<Attribute> atts,
		Coord gravity,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vel.size()) return;

		Attribute att = atts[pId];

		if (att.isDynamic())
		{
				vel[pId] += dt * (gravity);	
		}
	}

	template<typename Real, typename Coord>
	__global__ void K_Friction(
		DArray<Coord> Norm,
		DArray<Coord> Force,
		DArray<Coord> Velocity,
		Real mu,
		Coord g,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= Norm.size()) return;
		
		Coord f = Force[pId] + g;
		if (f.dot(Norm[pId]) <= EPSILON)//enforced
		{
			Coord f_n = f.dot(Norm[pId]) * Norm[pId];
			Coord f_t = f - f_n;
			Coord v_n = Velocity[pId].dot(Norm[pId]) * Norm[pId];
			Coord v_t = Velocity[pId] - v_n;
			if (v_t.norm() <= EPSILON)
				return;
			Real damp;
			if (f_t.norm() >= mu * f_n.norm()) //kinetic friction
			{
				damp = max(0.0, 1.0 - f_n.norm() * mu * dt / v_t.norm());
			}
			else { //static friction
				damp = 0.0;
			}
			Velocity[pId] = v_n + v_t * damp;
		}

	}

	template<typename Real, typename Coord>
	__global__ void K_Friction(
		DArray<Coord> Norm,
		DArray<Coord> Velocity,
		DArray<Attribute> atts,
		Real mu,
		Coord g,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= Norm.size()) return;

		Attribute att = atts[pId];

		if (att.isDynamic())
		{

			Coord f = g;
			if (f.dot(Norm[pId]) <= EPSILON)//enforced
			{
				Coord f_n = f.dot(Norm[pId]) * Norm[pId];
				Coord f_t = f - f_n;
				Coord v_n = Velocity[pId].dot(Norm[pId]) * Norm[pId];
				Coord v_t = Velocity[pId] - v_n;
				if (v_t.norm() <= EPSILON)
					return;
				Real damp;
				if (f_t.norm() >= mu * f_n.norm()) //kinetic friction
				{
					damp = max(0.0, 1.0 - f_n.norm() * mu * dt / v_t.norm());
				}
				else { //static friction
					damp = 0.0;
				}
				Velocity[pId] = v_n + v_t * damp;
			}
		}
	}


	template<typename TDataType>
	bool DamplingParticleIntegrator<TDataType>::updateVelocity()
	{
		Real dt = this->inTimeStep()->getData();

		auto scn = dyno::SceneGraphFactory::instance()->active();
		Coord gravity = scn->getGravity();

		int total_num = this->inPosition()->size();

		

		if (this->inAttribute()->isEmpty())
		{
			
			cuExecute(total_num,
				K_UpdateVelocity,
				this->inVelocity()->getData(),
				this->inContactForce()->getData(),
				this->inNorm()->getData(),
				gravity,
				dt);

			if (this->inMu()->getData() >= EPSILON) //Columbian friction model
			{
				cuExecute(total_num,
					K_Friction,
					this->inNorm()->getData(),
					this->inContactForce()->getData(),
					this->inVelocity()->getData(),
					this->inMu()->getData(),
					gravity,
					dt);
				

				cuSynchronize();
			}
		}
		else//with att
		{
			
			
			
			cuExecute(total_num,
				K_UpdateVelocity,
				this->inVelocity()->getData(),
				this->inPosition()->getData(),
				this->inContactForce()->getData(),
				this->inNorm()->getData(),
				this->inAttribute()->getData(),
				gravity,
				dt);

			if (this->inMu()->getData() >= EPSILON) //Columbian friction model
			{
				cuExecute(total_num,
					K_Friction,
					this->inNorm()->getData(),
					this->inVelocity()->getData(),
					this->inAttribute()->getData(),
					this->inMu()->getData(),
					gravity,
					dt);

				cuSynchronize();

			}
			
		}
		
		return true;
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DArray<Coord> pos,
		DArray<Coord> vel,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		pos[pId] += dt * vel[pId];
	}

	template<typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DArray<Coord> pos,
		DArray<Coord> vel,
		DArray<Attribute> att,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		Attribute att_i = att[pId];
		Coord po = pos[pId];
		if (!att_i.isFixed())
		{
			pos[pId] += dt * vel[pId];
		}
	}

	template<typename TDataType>
	bool DamplingParticleIntegrator<TDataType>::updatePosition()
	{
		Real dt = this->inTimeStep()->getData();

		int total_num = this->inPosition()->getDataPtr()->size();
		

		if (this->inAttribute()->isEmpty())
		{
			cuExecute(total_num,
				K_UpdatePosition,
				this->inPosition()->getData(),
				this->inVelocity()->getData(),
				dt);
		}
		else
		{
			cuExecute(total_num,
				K_UpdatePosition,
				this->inPosition()->getData(),
				this->inVelocity()->getData(),
				this->inAttribute()->getData(),
				dt);
		}


		return true;
	}

	template<typename TDataType>
	bool DamplingParticleIntegrator<TDataType>::integrate()
	{
		if (!this->inPosition()->isEmpty())
		{
			updateVelocity();
			updatePosition();
			
		}

		return true;
	}


	template<typename TDataType>
	void DamplingParticleIntegrator<TDataType>::compute()
	{
		this->begin();
		this->integrate();
		this->end();
	}



	DEFINE_CLASS(DamplingParticleIntegrator);
}