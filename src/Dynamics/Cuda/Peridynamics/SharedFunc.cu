#include "hip/hip_runtime.h"
#include "SharedFunc.h"

namespace dyno
{
	template <typename Coord, typename NPair>
	__global__ void K_UpdateRestShape(
		DArrayList<NPair> shape,
		DArrayList<int> nbr,
		DArray<Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		NPair np;

		List<NPair>& rest_shape_i = shape[pId];
		List<int>& list_id_i = nbr[pId];
		int nbSize = list_id_i.size();
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = list_id_i[ne];
			np.index = j;
			np.pos = pos[j];
			np.weight = 1;

			rest_shape_i.insert(np);
			if (pId == j)
			{
				NPair np_0 = rest_shape_i[0];
				rest_shape_i[0] = np;
				rest_shape_i[ne] = np_0;
			}
		}
	}


	template<typename Coord, typename NPair>
	void constructRestShape(DArrayList<NPair>& shape, DArrayList<int>& nbr, DArray<Coord>& pos)
	{
		cuExecute(nbr.size(),
			K_UpdateRestShape,
			shape,
			nbr,
			pos);
	}

	__global__ void K_AddOne(
		DArray<uint> num,
		DArrayList<int> nbr)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= nbr.size()) return;

		num[tId] = nbr[tId].size() + 1;
	}

	template <typename Coord, typename NPair>
	__global__ void K_UpdateRestShapeSelf(
		DArrayList<NPair> shape,
		DArray<Coord> pos)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= pos.size()) return;

		NPair np;
		np.index = tId;
		np.pos = pos[tId];
		np.weight = 1;

		shape[tId].insert(np);
	}

	template<typename Coord, typename NPair>
	void constructRestShapeWithSelf(DArrayList<NPair>& shape, DArrayList<int>& nbr, DArray<Coord>& pos)
	{
		DArray<uint> num(nbr.size());

		cuExecute(nbr.size(),
			K_AddOne,
			num,
			nbr);

		shape.resize(num);

		cuExecute(nbr.size(),
			K_UpdateRestShapeSelf,
			shape,
			pos);

		cuExecute(nbr.size(),
			K_UpdateRestShape,
			shape,
			nbr,
			pos);

		num.clear();
	}

	template void constructRestShape<Vec3f, TPair<DataType3f>>(DArrayList<TPair<DataType3f>>& shape, DArrayList<int>& nbr, DArray<Vec3f>& pos);
	template void constructRestShapeWithSelf<Vec3f, TPair<DataType3f>>(DArrayList<TPair<DataType3f>>& shape, DArrayList<int>& nbr, DArray<Vec3f>& pos);
}