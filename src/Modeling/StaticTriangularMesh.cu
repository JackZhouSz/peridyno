#include "hip/hip_runtime.h"
#include "StaticTriangularMesh.h"

#include "GLSurfaceVisualModule.h"

#include "Topology/TriangleSet.h"
#include <iostream>
#include <sys/stat.h>


namespace dyno
{
	IMPLEMENT_TCLASS(StaticTriangularMesh, TDataType)

	template<typename TDataType>
	StaticTriangularMesh<TDataType>::StaticTriangularMesh()
		: ParametricModel<TDataType>()
	{
		auto triSet = std::make_shared<TriangleSet<TDataType>>();
		this->stateTopology()->setDataPtr(triSet);
		this->outTriangleSet()->setDataPtr(triSet);

		this->inTriangleSet_IN()->setDataPtr(triSet);

		this->inTriangleSet_IN()->tagOptional(true);

		auto surfaceRender = std::make_shared<GLSurfaceVisualModule>();
		surfaceRender->setColor(Vec3f(0.8, 0.52, 0.25));
		surfaceRender->setVisible(true);
		this->stateTopology()->connect(surfaceRender->inTriangleSet());
		this->graphicsPipeline()->pushModule(surfaceRender);


	}

	template<typename TDataType>
	void StaticTriangularMesh<TDataType>::resetStates()
	{
		if (!(this->varConvertInput()->getData()))
		{
		
		auto triSet = TypeInfo::cast<TriangleSet<TDataType>>(this->stateTopology()->getDataPtr());
		
		if (this->varFileName()->getDataPtr()->string() == "")
			return;

		triSet->loadObjFile(this->varFileName()->getDataPtr()->string());

		triSet->scale(this->varScale()->getData());
		triSet->translate(this->varLocation()->getData());
		triSet->rotate(this->varRotation()->getData() * PI / 180);



		initPos.resize(triSet->getPoints().size());
		initPos.assign(triSet->getPoints());
		center = this->varCenter()->getData();
		centerInit = center;
		
		}
		else
		{
		auto intri = this->inTriangleSet_IN()->getDataPtr();
		this->outTriangleSet()->setDataPtr(std::make_shared<TriangleSet<TDataType>>());
		auto outtri = this->outTriangleSet()->getDataPtr();
		outtri->copyFrom(*intri);

		}

		Node::resetStates();
	}

	template <typename Coord, typename Matrix>
	__global__ void K_InitKernelFunctionMesh(
		DArray<Coord> posArr,
		DArray<Coord> posInit,
		Coord center,
		Coord centerInit,
		Matrix rotation
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size())
			return;
		Coord pos;
		pos = posInit[pId] - centerInit;
		pos = rotation * pos;
		posArr[pId] = pos + center;

	}


	template<typename TDataType>
	void StaticTriangularMesh<TDataType>::updateStates()
	{

		auto triSet = TypeInfo::cast<TriangleSet<TDataType>>(this->stateTopology()->getDataPtr());

		if (this->varSequence()->getData() == true)
		{


			std::string filename = this->varFileName()->getDataPtr()->string();
			int num_ = filename.rfind("_");

			filename.replace(num_ + 1, filename.length() - 4 - (num_ + 1), std::to_string(this->stateFrameNumber()->getData()));


			auto triSet = TypeInfo::cast<TriangleSet<TDataType>>(this->stateTopology()->getDataPtr());

			if (this->varSequence()->getData() == true)
			{


				std::string filename = this->varFileName()->getDataPtr()->string();
				int num_ = filename.rfind("_");

				filename.replace(num_ + 1, filename.length() - 4 - (num_ + 1), std::to_string(this->stateFrameNumber()->getData()));


				struct stat buffer;
				bool isvaid = (stat(filename.c_str(), &buffer) == 0);

				if (isvaid)
				{
					triSet->loadObjFile(filename);

					triSet->scale(this->varScale()->getData());
					triSet->translate(this->varLocation()->getData());
					triSet->rotate(this->varRotation()->getData() * PI / 180);

					initPos.resize(triSet->getPoints().size());
					initPos.assign(triSet->getPoints());
					center = this->varCenter()->getData();
					centerInit = center;
				}


			}


			Coord velocity = this->varVelocity()->getData();
			Coord angularVelocity = this->varAngularVelocity()->getData();

			//printf("velocity = %.10lf %.10lf %.10lf\n", velocity[0], velocity[1], velocity[2]);

			Real dt = 0.001f;
			rotQuat = rotQuat.normalize();
			rotQuat += dt * 0.5f *
				Quat<Real>(angularVelocity[0], angularVelocity[1], angularVelocity[2], 0.0) * (rotQuat);

			rotQuat = rotQuat.normalize();
			rotMat = rotQuat.toMatrix3x3();

			center += velocity * dt;

			cuExecute(triSet->getPoints().size(),
				K_InitKernelFunctionMesh,
				triSet->getPoints(),
				initPos,
				center,
				centerInit,
				rotMat
			);
		}
	}

	DEFINE_CLASS(StaticTriangularMesh);
}