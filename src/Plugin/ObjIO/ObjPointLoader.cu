#include "hip/hip_runtime.h"
#include "ObjPointLoader.h"

#include "Topology/TriangleSet.h"
#include <iostream>
#include <sys/stat.h>
#include "tinyobjloader/tiny_obj_loader.h"



namespace dyno
{
	IMPLEMENT_TCLASS(ObjPoint, TDataType)

		template<typename TDataType>
	ObjPoint<TDataType>::ObjPoint()
		: Node()
	{
		auto pointSet = std::make_shared<PointSet<TDataType>>();
		this->varRadius()->setRange(0,10);


		this->stateTopology()->setDataPtr(pointSet);

		this->outPointSet()->setDataPtr(pointSet);

		pointrender = new GLPointVisualModule;
		std::shared_ptr<GLPointVisualModule>ptrender(pointrender);
		
		pointrender->setVisible(true);
		pointrender->setColor(Color(1, 0, 0));
		pointrender->varPointSize()->setValue(this->varRadius()->getData());
		pointrender->varPointSize()->setValue(varRadius()->getData()/100);

		this->stateTopology()->connect(pointrender->inPointSet());
		this->graphicsPipeline()->pushModule(ptrender);

	}

	template<typename TDataType>
	void ObjPoint<TDataType>::resetStates()
	{
		auto pointSet = TypeInfo::cast<PointSet<TDataType>>(this->stateTopology()->getDataPtr());


		std::string filename = this->varFileName()->constDataPtr()->string();

		loadObj(*pointSet, filename);
		pointSet->scale(this->varScale()->getData());
		pointSet->translate(this->varLocation()->getData());
		pointSet->rotate(this->varRotation()->getData() * PI / 180);

		
		Node::resetStates();
		
		initPos.resize(pointSet->getPoints().size());
		initPos.assign(pointSet->getPoints());
		center = this->varCenter()->getData();
		centerInit = center;

		pointrender->varPointSize()->setValue(varRadius()->getData() / 100);

	}

	template <typename Coord, typename Matrix>
	__global__ void K_InitKernelFunctionMesh(
		DArray<Coord> posArr,
		DArray<Coord> posInit,
		Coord center,
		Coord centerInit,
		Matrix rotation
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size())
			return;
		Coord pos;
		pos = posInit[pId] - centerInit;
		pos = rotation * pos;
		posArr[pId] = pos + center;

	}

	
	template<typename TDataType>
	void ObjPoint<TDataType>::updateStates()
	{
		auto pointSet = TypeInfo::cast<PointSet<TDataType>>(this->stateTopology()->getDataPtr());

		if (this->varSequence()->getData() == true)
		{
			std::string filename = this->varFileName()->constDataPtr()->string();
			int num_ = filename.rfind("_");

			filename.replace(num_ + 1, filename.length() - 4 - (num_ + 1), std::to_string(this->stateFrameNumber()->getData()));

				loadObj(*pointSet,filename);
				pointSet->scale(this->varScale()->getData());
				pointSet->translate(this->varLocation()->getData());
				pointSet->rotate(this->varRotation()->getData() * PI / 180);

				initPos.resize(pointSet->getPoints().size());
				initPos.assign(pointSet->getPoints());
				center = this->varCenter()->getData();
				centerInit = center;
		}

		Coord velocity = this->varVelocity()->getData();
		Coord angularVelocity = this->varAngularVelocity()->getData();

		Real dt = 0.001f;
		rotQuat = rotQuat.normalize();
		rotQuat += dt * 0.5f *
			Quat<Real>(angularVelocity[0], angularVelocity[1], angularVelocity[2], 0.0) * (rotQuat);

		rotQuat = rotQuat.normalize();
		rotMat = rotQuat.toMatrix3x3();

		center += velocity * dt;

		if (!pointSet->getPointSize() == 0)
		{
			cuExecute(pointSet->getPoints().size(),
				K_InitKernelFunctionMesh,
				pointSet->getPoints(),
				initPos,
				center,
				centerInit,
				rotMat
			);
		}


	}


	template<typename TDataType>
	void ObjPoint<TDataType>::loadObj(PointSet<TDataType>& pointset, std::string filename)
	{
		std::vector<Coord> vertList;
		std::vector<TopologyModule::Triangle> faceList;

		tinyobj::attrib_t myattrib;
		std::vector <tinyobj::shape_t> myshape;
		std::vector <tinyobj::material_t> mymat;
		std::string mywarn;
		std::string myerr;

		char* fname = (char*)filename.c_str();
		std::cout << fname << std::endl;
		tinyobj::LoadObj(&myattrib, &myshape, &mymat, &mywarn, &myerr, fname, nullptr, true, true);
		std::cout << mywarn << std::endl;
		std::cout << myerr << std::endl;
		std::cout << "************************    Loading : shapelod    ************************  " << std::endl << std::endl;
		std::cout << "                        " << "    shape size =" << myshape.size() << std::endl << std::endl;
		std::cout << "************************    Loading : v    ************************  " << std::endl << std::endl;
		std::cout << "                        " << "    point sizelod = " << myattrib.GetVertices().size() / 3 << std::endl << std::endl;

		if (myshape.size() == 0) 
		{ 
			if (myattrib.GetVertices().size() == 0)
			{
				return;
			}
		}

		for (int i = 0; i < myattrib.GetVertices().size() / 3; i++)
		{

			vertList.push_back(Coord(myattrib.GetVertices()[3 * i], myattrib.GetVertices()[3 * i + 1], myattrib.GetVertices()[3 * i + 2]));
		}
		std::cout << "************************    Loading : f    ************************  " << std::endl << std::endl;
		for (int i = 0; i < myshape.size(); i++)
		{
			std::cout << "                        " << "    Triangle " << i << " size =" << myshape[i].mesh.indices.size() / 3 << std::endl << std::endl;

			for (int s = 0; s < myshape[i].mesh.indices.size() / 3; s++)
			{
				//std::cout << myshape[i].mesh.indices[s].vertex_index <<"  " << std::endl;

				faceList.push_back(TopologyModule::Triangle(myshape[i].mesh.indices[3 * s].vertex_index, myshape[i].mesh.indices[3 * s + 1].vertex_index, myshape[i].mesh.indices[3 * s + 2].vertex_index));
			}
		}
		std::cout << "************************    Loading completed    **********************" << std::endl << std::endl;
		int s = vertList.size();
		printf("%d", s);
		pointset.setPoints(vertList);
	}

	template<typename TDataType>
	void ObjPoint<TDataType>::convertData(TriangleSet<TDataType>& Triangleset, std::string filename, PointSet<TDataType>& PointSet)
	{

	}

	DEFINE_CLASS(ObjPoint);
}